#include "hip/hip_runtime.h"

#include "cuda_vnd.h"

#include <stdio.h>

__global__ void kernel_vnd_test( int c1, int c2 )
{
    c1++;
    c2++;
    printf( "? %d \n", ( c1 + c2 ) );
}

__global__ void kernel_compute_best_vnd( float cRadius,
                                         CCircle* cCircles, int cNumCircles,
                                         CPair* cPairs, int cNumPairs,
                                         CCircle* cCirclesExt )
{
    int tIndx = threadIdx.x;
    if ( tIndx < 0 || tIndx >= cNumPairs )
    {
        return;
    }

    int c1 = cPairs[tIndx].c1;
    int c2 = cPairs[tIndx].c2;

    // Make the swap 
    int windxOff =  sizeof( CCircle ) * cNumCircles * tIndx;
    swap_circles( c1, c2, windxOff, cCirclesExt );

    // Intensify this new configuration

}

__device__ void swap_circles( int c1, int c2, int windxOff, CCircle* cCirclesExt )
{
    float _x = cCirclesExt[windxOff + c1].x;
    float _y = cCirclesExt[windxOff + c1].y;

    cCirclesExt[windxOff + c1].x = cCirclesExt[windxOff + c2].x;
    cCirclesExt[windxOff + c1].y = cCirclesExt[windxOff + c2].y;

    cCirclesExt[windxOff + c2].x = _x;
    cCirclesExt[windxOff + c2].y = _y;
}

__device__ void optimize( int windxOff, CCircle* cCirclesExt )
{
    
}

extern "C" 
{


    void computeVND( float cRadius, 
                     CCircle* circles, int numCircles, 
                     CPair* pairs, int numPairs,
                     int *h_bestIndx )
    {
        CCircle* d_circles;
        CPair* d_pairs;

        hipMalloc( &d_circles, sizeof( CCircle ) * numCircles );
        hipMemcpy( d_circles, circles, 
                    sizeof( CCircle ) * numCircles, 
                    hipMemcpyHostToDevice );

        hipMalloc( &d_pairs, sizeof( CPair ) * numPairs );
        hipMemcpy( d_pairs, pairs, 
                    sizeof( CPair ) * numPairs, 
                    hipMemcpyHostToDevice );

        printf( "starting kernel\n" );

        // Create the space neccessary for each thread to work in
        CCircle* d_circlesExt;
        hipMalloc( &d_circlesExt, sizeof( CCircle ) * numCircles * numPairs );
        for ( int q = 0; q < numPairs; q++ )
        {
            hipMemcpy( d_circlesExt + sizeof( CCircle ) * numCircles * q , circles, sizeof( CCircle ) * numCircles, hipMemcpyHostToDevice );
        }

        //kernel_vnd_test<<<1, numPairs>>>( 1, 2 );
        kernel_compute_best_vnd<<<1, numPairs>>>( cRadius, d_circles, numCircles, d_pairs, numPairs, d_circlesExt );
    }
}