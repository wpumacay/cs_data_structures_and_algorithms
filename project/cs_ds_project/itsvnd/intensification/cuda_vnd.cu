#include "hip/hip_runtime.h"

#include "cuda_vnd.h"

#include <stdio.h>

#define K_OPTIMIZER_ITERATIONS 100
#define K_GRAD_STEP 0.1f
#define K_DELTA_STEP 0.01f
#define K_RANGE_U_FEASIBLE 1
#define K_R_V 0.25

__device__ void k_swap_circles( int c1, int c2, int wIndxOff, CCircle* cCirclesExt )
{
    float _x = cCirclesExt[wIndxOff + c1].x;
    float _y = cCirclesExt[wIndxOff + c1].y;

    cCirclesExt[wIndxOff + c1].x = cCirclesExt[wIndxOff + c2].x;
    cCirclesExt[wIndxOff + c1].y = cCirclesExt[wIndxOff + c2].y;

    cCirclesExt[wIndxOff + c2].x = _x;
    cCirclesExt[wIndxOff + c2].y = _y;
}

__device__ float k_potential( CVector &xConf,
                              CVector &rConf )
{
    float _res = 0.0f;

    float _rContainer = xConf[0];
    // Overlap with the container
    for ( int q = 0; q < rConf.dim(); q++ )
    {
        float _xi = xConf[ 2 * q + 1 ];
        float _yi = xConf[ 2 * q + 2 ];
        float _ri = rConf[q];
        float _dist = sqrtf( _xi * _xi + _yi * _yi );
        float _cost_q = fmaxf( _dist + _ri - _rContainer, 0.0f );

        _res += _cost_q * _cost_q;
    }
    // Overlap between circles
    for ( int q = 0; q < rConf.dim(); q++ )
    {
        float _xi = xConf[ 2 * q + 1 ];
        float _yi = xConf[ 2 * q + 2 ];
        float _ri = rConf[q];

        for ( int p = q + 1; p < rConf.dim(); p++ )
        {
            float _xj = xConf[ 2 * p + 1 ];
            float _yj = xConf[ 2 * p + 2 ];
            float _rj = rConf[p];

            float _dist = sqrtf( ( _xi - _xj ) * ( _xi - _xj ) + 
                                 ( _yi - _yj ) * ( _yi - _yj ) );

            float _cost_pq = fmaxf( _ri + _rj - _dist, 0.0f );

            _res += _cost_pq * _cost_pq;
        }
    }

    return _res;
}

__device__ CVector k_computeGradient( CVector &xConf,
                                      CVector &rConf )
{
    CVector _grad( xConf.dim() );
    float _step = K_DELTA_STEP;

    float _f = k_potential( xConf, rConf );

    for ( int q = 0; q < xConf.dim(); q++ )
    {
        CVector _xc = xConf;
        _xc[q] += _step;
        float _f_next = k_potential( _xc, rConf );
        _grad[q] = ( _f_next - _f ) / _step;
    }

    return _grad;
}

__device__ float k_container_potential( CVector &xConf,
                                        CVector &rConf )
{
    float _res = 0.0f;

    float _rContainer = xConf[0];
    // Overlap with the container
    for ( int q = 0; q < rConf.dim(); q++ )
    {
        float _xi = xConf[ 2 * q + 1 ];
        float _yi = xConf[ 2 * q + 2 ];
        float _ri = rConf[q];
        float _dist = sqrtf( _xi * _xi + _yi * _yi );
        float _cost_q = fmaxf( _dist + _ri - _rContainer, 0.0f );

        _res += _cost_q * _cost_q;
    }

    return _res;
}

__device__ float k_computeContainerFeasibility( int wIndxOff, CCircle* cCirclesExt, float cRadius, int cNumCircles )
{
    CVector _xConf( 1 + 2 * cNumCircles );
    CVector _rConf( cNumCircles );

    _xConf[0] = cRadius;
    for ( int q = 0; q < cNumCircles; q++ )
    {
        _xConf[ 1 + 2 * q ] = cCirclesExt[wIndxOff + q].x;
        _xConf[ 2 + 2 * q ] = cCirclesExt[wIndxOff + q].y;
        _rConf[q] = cCirclesExt[wIndxOff + q].r;
    }

    return k_container_potential( _xConf, _rConf );
}

__device__ void k_optimize( int wIndxOff, CCircle* cCirclesExt, float& cRadius, int cNumCircles )
{

    for ( int s = 0; s < K_OPTIMIZER_ITERATIONS; s++ )
    {
        CVector _xConf( 1 + 2 * cNumCircles );
        CVector _rConf( cNumCircles );

        _xConf[0] = cRadius;
        for ( int q = 0; q < cNumCircles; q++ )
        {
            _xConf[ 1 + 2 * q ] = cCirclesExt[wIndxOff + q].x;
            _xConf[ 2 + 2 * q ] = cCirclesExt[wIndxOff + q].y;
            _rConf[q] = cCirclesExt[wIndxOff + q].r;
        }
        // Perform a step
        CVector _grad = k_computeGradient( _xConf, _rConf );
        _xConf = _xConf - _grad * K_GRAD_STEP;

        cRadius = _xConf[0];

        for ( int q = 0; q < cNumCircles; q++ )
        {
            cCirclesExt[wIndxOff + q].x = _xConf[ 1 + 2 * q ];
            cCirclesExt[wIndxOff + q].y = _xConf[ 2 + 2 * q ];
        }
    }
}

__global__ void kernel_compute_best_vnd( float cRadius,
                                         CCircle* cCircles, int cNumCircles,
                                         CPair* cPairs, int cNumPairs,
                                         CCircle* cCirclesExt,
                                         int* d_bests, float* d_bestsRadius )
{
    int tIndx = threadIdx.x;
    if ( tIndx < 0 || tIndx >= cNumPairs )
    {
        return;
    }

    int c1 = cPairs[tIndx].c1;
    int c2 = cPairs[tIndx].c2;

    // Make the swap 
    int wIndxOff =  cNumCircles * tIndx;
    k_swap_circles( c1, c2, wIndxOff, cCirclesExt );

    float containerRadius = cRadius;

    // Intensify this new configuration *************************************************
    k_optimize( wIndxOff, cCirclesExt, containerRadius, cNumCircles );

    // Check if touches container too much
    float _cFeasibility = k_computeContainerFeasibility( wIndxOff, cCirclesExt, cRadius, cNumCircles );

    // **********************************************************************************

    if ( _cFeasibility < K_RANGE_U_FEASIBLE )
    {
        // binary search
        
        int k = containerRadius / K_R_V;
        int _upBound = k - 1;
        int _loBound = 0;
        float _sum;
        for ( int q = 0; q < cNumCircles; q++ )
        {
            float _frac = cCirclesExt[wIndxOff + q].r / K_R_V;
            _sum += _frac * _frac;
        }
        _loBound = floorf( sqrtf( _sum ) );

        do
        {
            float _contFeasib = k_computeContainerFeasibility( wIndxOff, cCirclesExt, cRadius, cNumCircles );
            if ( _contFeasib < K_RANGE_U_FEASIBLE )
            {
                _upBound = k - 1;
            }
            else
            {
                _loBound = k;
            }
            k = ( _loBound + _upBound ) / 2;
            containerRadius = k * K_R_V;
            float _copyContRadius = containerRadius;
            k_optimize( wIndxOff, cCirclesExt, _copyContRadius, cNumCircles );

            if ( _upBound - _loBound <= 1 && k == _loBound )
            {
                break;
            }

        }while( _upBound > _loBound );
    }

    // Check if is better than the current solution

    float _best_r = cRadius;

    if ( containerRadius < _best_r )
    {
        if ( fabsf( containerRadius - _best_r ) > 0.5 )
        {
            d_bests[tIndx] = tIndx;
            d_bestsRadius[tIndx] = containerRadius;
        }
    }
}


bool computeVND( float& cRadius, 
                 CCircle* circles, int numCircles, 
                 CPair* pairs, int numPairs )
{

    // Reserve space for the current configuration and  ****
    // possible swap moves
    CCircle* d_circles;
    CPair* d_pairs;

    hipMalloc( ( void** ) &d_circles, sizeof( CCircle ) * numCircles );
    hipMemcpy( d_circles, circles, 
                sizeof( CCircle ) * numCircles, 
                hipMemcpyHostToDevice );

    hipMalloc( ( void** ) &d_pairs, sizeof( CPair ) * numPairs );
    hipMemcpy( d_pairs, pairs, 
                sizeof( CPair ) * numPairs, 
                hipMemcpyHostToDevice );
    // *****************************************************

    // Create arrays to store the results ***********

    int* h_bests = new int[numPairs];
    for ( int q = 0; q < numPairs; q++ )
    {
        h_bests[q] = -1;
    }

    int* d_bests;
    hipMalloc( ( void** ) &d_bests, sizeof( int ) * numPairs );
    hipMemcpy( d_bests, h_bests, sizeof( int ) * numPairs, hipMemcpyHostToDevice );

    float* h_bestsRadius = new float[numPairs];
    for ( int q = 0; q < numPairs; q++ )
    {
        h_bestsRadius[q] = 0.0f;
    }

    float* d_bestsRadius;
    hipMalloc( ( void** ) &d_bestsRadius, sizeof( float ) * numPairs );
    hipMemcpy( d_bestsRadius, h_bestsRadius, sizeof( float ) * numPairs, hipMemcpyHostToDevice );

    // **********************************************

    // Create the space neccessary for each thread to work in ****************

    CCircle* h_circlesExt = new CCircle[numCircles * numPairs];
    for ( int q = 0; q < numPairs; q++ )
    {
        for ( int p = 0; p < numCircles; p++ )
        {
            h_circlesExt[q * numCircles + p].r = circles[p].r;
            h_circlesExt[q * numCircles + p].x = circles[p].x;
            h_circlesExt[q * numCircles + p].y = circles[p].y;
        }
    }

    CCircle* d_circlesExt;
    hipMalloc( ( void** ) &d_circlesExt, sizeof( CCircle ) * numCircles * numPairs );
    hipMemcpy( d_circlesExt, h_circlesExt, sizeof( CCircle ) * numCircles * numPairs, hipMemcpyHostToDevice );

    // ***********************************************************************
    
    kernel_compute_best_vnd<<<1, numPairs>>>( cRadius, 
                                              d_circles, numCircles, 
                                              d_pairs, numPairs, 
                                              d_circlesExt, d_bests, d_bestsRadius );
    
    // Retrieve the search results ********************

    hipMemcpy( h_bests, d_bests, sizeof( int ) * numPairs, hipMemcpyDeviceToHost );
    hipMemcpy( h_bestsRadius, d_bestsRadius, sizeof( float ) * numPairs, hipMemcpyDeviceToHost );
    hipMemcpy( h_circlesExt, d_circlesExt, sizeof( CCircle ) * numCircles * numPairs, hipMemcpyDeviceToHost );

    // ************************************************

    // for all search results that gave a found-better solution ...
    // get the best among them 

    int _bestIndx = -1;
    float _bestRadius = 1000000.0f;


    for ( int q = 0; q < numPairs; q++ )
    {
        if ( h_bests[q] == -1 )
        {
            continue;
        }

        if ( h_bestsRadius[q] < _bestRadius )
        {
            _bestIndx = q;
            _bestRadius = h_bestsRadius[q];
        }
    }

    bool foundBetter = false;

    if ( _bestIndx != -1 )
    {

        printf( "found better \n" );
        foundBetter = true;
        // If a better solution was found, use this to ...
        // update the circle configuration

        int wIndxOff = numCircles * _bestIndx;

        cRadius = _bestRadius;

        for ( int q = 0; q < numCircles; q++ )
        {
            
            circles[q].x = h_circlesExt[wIndxOff + q].x;
            circles[q].y = h_circlesExt[wIndxOff + q].y;
            circles[q].r = h_circlesExt[wIndxOff + q].r;
        }
    }

    hipFree( d_circles );
    hipFree( d_pairs );
    hipFree( d_bests );
    hipFree( d_bestsRadius );
    hipFree( d_circlesExt );

    return foundBetter;
}
