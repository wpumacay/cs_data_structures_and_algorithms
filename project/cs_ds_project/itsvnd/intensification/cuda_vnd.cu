#include "hip/hip_runtime.h"

extern "C" 
{
    
    #include "cuda_vnd.h"

}


__global__ void kernel_vnd_test( int c1, int c2 )
{
    c1++;
    c2++;
    printf( "? %d \n", ( c1 + c2 ) );
}



extern "C" 
{


    void computeVND( float cRadius, 
                                CCircle* circles, int numCircles, 
                                CPair* pairs, int numPairs,
                                int *h_bestIndx )
    {
        CCircle* d_circles;
        CPair* d_pairs;

        hipMalloc( &d_circles, sizeof( CCircle ) * numCircles );
        hipMalloc( &d_pairs, sizeof( CPair ) * numPairs );
        hipMemcpy( d_circles, circles, 
                    sizeof( CCircle ) * numCircles, 
                    hipMemcpyHostToDevice );
        hipMemcpy( d_pairs, pairs, 
                    sizeof( CPair ) * numPairs, 
                    hipMemcpyHostToDevice );

        kernel_vnd_test<<<1, numPairs>>>( 1, 2 );
    }
}