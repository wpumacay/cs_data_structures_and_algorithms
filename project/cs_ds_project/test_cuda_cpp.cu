
#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>

#define RANDOM() ( rand() / ( float )RAND_MAX )

#define WX 100.0
#define WY 100.0
#define WV 2.0
#define N_PARTICLES 100
#define PI 3.1415926535

class CVector
{

    private :

    float* m_buff;
    int m_dim;

    public :

    __host__ __device__ CVector()
    {
        m_dim = 0;
        m_buff = NULL;
    }

    __host__ __device__ CVector( int dim )
    {
        m_dim = dim;
        m_buff = ( float* ) malloc( sizeof( float ) * m_dim );
        for ( int q = 0; q < m_dim; q++ )
        {
            m_buff[q] = 0.0f;
        }
    }

    __host__ __device__ CVector( const CVector& vOther )
    {
        m_dim = vOther.m_dim;

        m_buff = ( float* ) malloc( sizeof( float ) * m_dim );
        for ( int q = 0; q < m_dim; q++ )
        {
            m_buff[q] = vOther.m_buff[q];
        }
    }

    __host__ __device__ ~CVector()
    {
        if ( m_buff != NULL )
        {
            free( m_buff );
        }
        m_buff = NULL;
    }

    __host__ __device__ CVector operator= ( const CVector& vOther )
    {
        if ( m_buff )
        {
            free( m_buff );
            m_buff = NULL;
        }
        
        m_dim = vOther.m_dim;

        m_buff = ( float* ) malloc( sizeof( float ) * m_dim );
        for ( int q = 0; q < m_dim; q++ )
        {
            m_buff[q] = vOther.m_buff[q];
        }
       
        return *this;
    }

    __host__ __device__ void set( int indx, float value )
    {
        m_buff[indx] = value;
    }

    __host__ __device__ int dim()
    {
        return m_dim;
    }

    __host__ __device__ float& operator[]( int indx )
    {
        return m_buff[indx];
    }

    __host__ __device__ CVector operator+ ( const CVector& other )
    {
        CVector _res( m_dim );
        
        if ( other.m_dim != m_dim )
        {
            return _res;
        }

        for ( int q = 0; q < m_dim; q++ )
        {
            _res[q] = m_buff[q] + other.m_buff[q];
        }

        return _res;
    }

    __host__ __device__ CVector operator- ( const CVector& other )
    {
        CVector _res( m_dim );
        
        if ( other.m_dim != m_dim )
        {
            return _res;
        }

        for ( int q = 0; q < m_dim; q++ )
        {
            _res[q] = m_buff[q] - other.m_buff[q];
        }

        return _res;
    }

    __host__ __device__ CVector operator* ( float scale )
    {
        for ( int q = 0; q < m_dim; q++ )
        {
            m_buff[q] *= scale;
        }

        return *this;
    }
};


class CParticle
{

    public :

    CVector xy;
    CVector v;

    __host__ __device__ CParticle()
    {
        xy = CVector( 2 );
        v = CVector( 2 );
    }

    __host__ __device__ CParticle( float x, float y,
                                   float vx, float vy )
    {
        xy = CVector( 2 );
        xy[0] = x;
        xy[1] = y;

        v = CVector( 2 );
        v[0] = vx;
        v[1] = vy;
    }
    
    __host__ __device__ CParticle( const CParticle& pOther )
    {
        xy = pOther.xy;
        v  = pOther.v;
    }

    __host__ __device__ CParticle operator= ( const CParticle& pOther )
    {
        xy = pOther.xy;
        v  = pOther.v;

        return *this;
    }

    __host__ __device__ void step( float dt )
    {
        xy = xy + v * dt;
    }
    
};

__global__ void kernel_updateParticles( CParticle* cParticles, int cNumParticles, float dt )
{
    int tIndx = threadIdx.x;
    
    if ( tIndx >= cNumParticles )
    {
        return;
    }

    cParticles[tIndx].step( dt );
}


int main()
{

    CParticle* h_particles = new CParticle[N_PARTICLES];
    for ( int q = 0; q < N_PARTICLES; q++ )
    {
        h_particles[q].xy[0] = RANDOM() * WX;
        h_particles[q].xy[1] = RANDOM() * WY;

        float _ang = RANDOM() * 2 * PI;

        h_particles[q].v[0] = WV * cos( _ang );
        h_particles[q].v[1] = WV * sin( _ang );
    }

    CParticle* d_particles;
    hipMalloc( ( void** ) &d_particles, sizeof( CParticle ) * N_PARTICLES );
    hipMemcpy( d_particles, h_particles, sizeof( CParticle ) * N_PARTICLES, hipMemcpyHostToDevice );


    // Just loop for now
    float T = 100.0;
    float dt = 0.01;
    int nIters = ( int ) ( T / dt );
    for ( int q = 0; q < nIters; q++ )
    {
        kernel_updateParticles<<<1,N_PARTICLES>>>( d_particles, N_PARTICLES, dt );
    }

    delete[] h_particles;
    hipFree( d_particles );

    return 0;
}



